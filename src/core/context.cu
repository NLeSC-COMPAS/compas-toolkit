#include <hip/hip_runtime.h>

#include <iostream>
#include <stdexcept>
#include <utility>
#include <cstring>

#include "context.h"

namespace compas {

static std::string format_exception_message(hipError_t err, const char* file, const int line) {
    const char* name = "";
    const char* msg = "";

    hipDrvGetErrorName(err, &name);
    hipDrvGetErrorString(err, &msg);

    char output[1024];
    snprintf(output, sizeof output, "CUDA error: %s (%s) at %s:%d", name, msg, file, line);

    return output;
}

static std::string format_exception_message(hipError_t err, const char* file, const int line) {
    auto name = hipGetErrorName(err);
    auto msg = hipGetErrorString(err);

    char output[1024];
    snprintf(output, sizeof output, "CUDA error: %s (%s) at %s:%d", name, msg, file, line);

    return output;
}

CudaException::CudaException(hipError_t err, const char* file, const int line) :
    message_(format_exception_message(err, file, line)) {
    //
}

CudaException::CudaException(hipError_t err, const char* file, const int line) :
    message_(format_exception_message(err, file, line)) {
    //
}

CudaException::CudaException(std::string msg) : message_("CUDA error: " + msg) {
    //
}

struct CudaContextImpl {
    CudaContextImpl(hipDevice_t device) {
        COMPAS_CUDA_CHECK(hipInit(0));
        COMPAS_CUDA_CHECK(hipDevicePrimaryCtxRetain(&context, device));
    }

    ~CudaContextImpl() {
        try {
            COMPAS_CUDA_CHECK(hipDevicePrimaryCtxRelease(device));
        } catch (const CudaException& e) {
            std::cerr << "ignoring cuda error: " << e.what() << "\n";
        }
    }

    hipDevice_t device = 0;
    hipCtx_t context = nullptr;
};

CudaContext make_context(int device) {
    return std::make_shared<CudaContextImpl>(device);
}

CudaContextGuard::CudaContextGuard(std::shared_ptr<CudaContextImpl> impl) : impl_(std::move(impl)) {
    COMPAS_CUDA_CHECK(hipCtxPushCurrent(impl_->context));
}

CudaContextGuard::~CudaContextGuard() noexcept(false) {
    // Synchronize first, then pop the context, then check the synchronization result. This way, the current
    // context will be popped even if `hipCtxSynchronize` returns an error.
    auto synchronize_result = hipCtxSynchronize();

    try {
        hipCtx_t current;
        COMPAS_CUDA_CHECK(hipCtxPopCurrent(&current));
    } catch (const CudaException& e) {
        std::cerr << "ignoring cuda error: " << e.what() << "\n";
    }

    COMPAS_CUDA_CHECK(synchronize_result);
}

std::string CudaContext::device_name() const {
    CudaContextGuard guard {*this};
    char name[512] = {0};
    COMPAS_CUDA_CHECK(hipDeviceGetName(name, sizeof(name), impl_->device));
    return name;
}

std::shared_ptr<CudaBuffer> CudaContext::allocate_buffer(size_t nbytes) const {
    return std::make_shared<CudaBuffer>(*this, nbytes);
}

void CudaContext::fill_buffer(
    hipDeviceptr_t output_ptr,
    size_t num_elements,
    const void* fill_value,
    size_t element_nbytes) const {
    COMPAS_ASSERT(element_nbytes > 0);
    size_t nbytes = num_elements * element_nbytes;

    bool all_equal = true;
    for (size_t i = 1; i < element_nbytes; i++) {
        if (static_cast<const char*>(fill_value)[i] != static_cast<const char*>(fill_value)[0]) {
            all_equal = false;
        }
    }

    CudaContextGuard guard {*this};

    if (all_equal || element_nbytes == 1) {
        char value = static_cast<const char*>(fill_value)[0];
        COMPAS_CUDA_CHECK(hipMemsetD8(output_ptr, value, nbytes));
    } else if (element_nbytes == 2) {
        uint16_t value;
        std::memcpy(&value, fill_value, element_nbytes);
        COMPAS_CUDA_CHECK(hipMemsetD16(output_ptr, value, nbytes));
    } else if (element_nbytes == 4) {
        uint32_t value;
        std::memcpy(&value, fill_value, element_nbytes);
        COMPAS_CUDA_CHECK(hipMemsetD32(output_ptr, value, nbytes));
    } else {
        COMPAS_PANIC("fill can only be performed using 8, 16, or 32 bit values");
    }
}

CudaBuffer::CudaBuffer(const CudaContext& context, hipDeviceptr_t ptr, size_t nbytes) :
    context_(context),
    is_owned_(false),
    device_ptr_(ptr),
    nbytes_(nbytes) {}

CudaBuffer::CudaBuffer(const CudaContext& context, size_t nbytes) :
    context_(context),
    is_owned_(true),
    device_ptr_(hipDeviceptr_t {}),
    nbytes_(0) {
    if (nbytes > 0) {
        CudaContextGuard guard {context_};
        COMPAS_CUDA_CHECK(hipMalloc((hipDeviceptr_t*)&device_ptr_, nbytes));
        nbytes_ = nbytes;
    }
}

CudaBuffer::~CudaBuffer() {
    if (nbytes_ > 0 && is_owned_) {
        try {
            CudaContextGuard guard {context_};
            COMPAS_CUDA_CHECK(hipFree(device_ptr_));
        } catch (const CudaException& e) {
            std::cerr << "ignoring cuda error: " << e.what() << "\n";
        }
    }
}

void CudaBuffer::copy_from_host(const void* host_ptr, size_t offset, size_t length) {
    COMPAS_ASSERT(offset <= nbytes_ && length <= nbytes_ - offset);

    CudaContextGuard guard {context_};
    COMPAS_CUDA_CHECK(hipMemcpyHtoD(device_ptr_ + offset, host_ptr, length));
}

void CudaBuffer::copy_to_host(void* host_ptr, size_t offset, size_t length) {
    COMPAS_ASSERT(offset <= nbytes_ && length <= nbytes_ - offset);

    CudaContextGuard guard {context_};
    COMPAS_CUDA_CHECK(hipMemcpyDtoH(host_ptr, device_ptr_ + offset, length));
}

void CudaBuffer::copy_from_device(hipDeviceptr_t src_ptr, size_t offset, size_t length) {
    COMPAS_ASSERT(offset <= nbytes_ && length <= nbytes_ - offset);

    CudaContextGuard guard {context_};
    COMPAS_CUDA_CHECK(hipMemcpyDtoD(src_ptr, device_ptr_ + offset, length));
}

void CudaBuffer::copy_to_device(hipDeviceptr_t dst_ptr, size_t offset, size_t length) {
    COMPAS_ASSERT(offset <= nbytes_ && length <= nbytes_ - offset);

    CudaContextGuard guard {context_};
    COMPAS_CUDA_CHECK(hipMemcpyDtoD(device_ptr_ + offset, dst_ptr, length));
}

}  // namespace compas