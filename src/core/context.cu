#include <hip/hip_runtime.h>

#include <iostream>
#include <stdexcept>
#include <utility>

#include "context.h"

namespace compas {

static std::string format_exception_message(hipError_t err, const char* file, const int line) {
    const char* name = "";
    const char* msg = "";

    hipDrvGetErrorName(err, &name);
    hipDrvGetErrorString(err, &msg);

    char output[1024];
    snprintf(output, sizeof output, "CUDA error: %s (%s) at %s:%d", name, msg, file, line);

    return output;
}

static std::string format_exception_message(hipError_t err, const char* file, const int line) {
    auto name = hipGetErrorName(err);
    auto msg = hipGetErrorString(err);

    char output[1024];
    snprintf(output, sizeof output, "CUDA error: %s (%s) at %s:%d", name, msg, file, line);

    return output;
}

CudaException::CudaException(hipError_t err, const char* file, const int line) :
    message_(format_exception_message(err, file, line)) {
    //
}

CudaException::CudaException(hipError_t err, const char* file, const int line) :
    message_(format_exception_message(err, file, line)) {
    //
}

CudaException::CudaException(std::string msg) : message_("CUDA error: " + msg) {
    //
}

struct CudaContextImpl {
    CudaContextImpl(hipDevice_t device) {
        COMPAS_CUDA_CHECK(hipInit(0));
        COMPAS_CUDA_CHECK(hipDevicePrimaryCtxRetain(&context, device));
    }

    ~CudaContextImpl() {
        try {
            COMPAS_CUDA_CHECK(hipDevicePrimaryCtxRelease(device));
        } catch (const CudaException& e) {
            std::cerr << "ignoring cuda error: " << e.what() << "\n";
        }
    }

    hipDevice_t device = 0;
    hipCtx_t context = nullptr;
};

CudaContext make_context(int device) {
    return std::make_shared<CudaContextImpl>(device);
}

CudaContextGuard::CudaContextGuard(std::shared_ptr<CudaContextImpl> impl) : impl_(std::move(impl)) {
    COMPAS_CUDA_CHECK(hipCtxPushCurrent(impl_->context));
}

CudaContextGuard::~CudaContextGuard() {
    try {
        hipCtx_t current;
        COMPAS_CUDA_CHECK(hipCtxPopCurrent(&current));
    } catch (const CudaException& e) {
        std::cerr << "ignoring cuda error: " << e.what() << "\n";
    }
}

std::string CudaContext::device_name() const {
    CudaContextGuard guard {*this};
    char name[512] = {0};
    COMPAS_CUDA_CHECK(hipDeviceGetName(name, sizeof(name), impl_->device));
    return name;
}

std::shared_ptr<CudaBuffer> CudaContext::allocate_buffer(size_t nbytes) const {
    return std::make_shared<CudaBuffer>(*this, nbytes);
}

CudaBuffer::CudaBuffer(const CudaContext& context, hipDeviceptr_t ptr, size_t nbytes) :
    context_(context),
    is_owned_(false),
    device_ptr_(ptr),
    nbytes_(nbytes) {}

CudaBuffer::CudaBuffer(const CudaContext& context, size_t nbytes) :
    context_(context),
    is_owned_(true),
    device_ptr_(hipDeviceptr_t {}),
    nbytes_(0) {
    if (nbytes > 0) {
        CudaContextGuard guard {context_};
        COMPAS_CUDA_CHECK(hipMalloc((hipDeviceptr_t*)&device_ptr_, nbytes));
        nbytes_ = nbytes;
    }
}

CudaBuffer::~CudaBuffer() {
    if (nbytes_ > 0 && is_owned_) {
        try {
            CudaContextGuard guard {context_};
            COMPAS_CUDA_CHECK(hipFree((hipDeviceptr_t)device_ptr_));
        } catch (const CudaException& e) {
            std::cerr << "ignoring cuda error: " << e.what() << "\n";
        }
    }
}

void CudaBuffer::copy_from_host(const void* host_ptr, size_t offset, size_t length) {
    COMPAS_ASSERT(offset <= nbytes_ && length <= nbytes_ - offset);

    CudaContextGuard guard {context_};
    COMPAS_CUDA_CHECK(hipMemcpyHtoD(device_ptr_ + offset, host_ptr, length));
}

void CudaBuffer::copy_to_host(void* host_ptr, size_t offset, size_t length) {
    COMPAS_ASSERT(offset <= nbytes_ && length <= nbytes_ - offset);

    CudaContextGuard guard {context_};
    COMPAS_CUDA_CHECK(hipMemcpyDtoH(host_ptr, device_ptr_ + offset, length));
}

void CudaBuffer::copy_from_device(hipDeviceptr_t src_ptr, size_t offset, size_t length) {
    COMPAS_ASSERT(offset <= nbytes_ && length <= nbytes_ - offset);

    CudaContextGuard guard {context_};
    COMPAS_CUDA_CHECK(hipMemcpyDtoD(src_ptr, device_ptr_ + offset, length));
}

void CudaBuffer::copy_to_device(hipDeviceptr_t dst_ptr, size_t offset, size_t length) {
    COMPAS_ASSERT(offset <= nbytes_ && length <= nbytes_ - offset);

    CudaContextGuard guard {context_};
    COMPAS_CUDA_CHECK(hipMemcpyDtoD(device_ptr_ + offset, dst_ptr, length));
}

void CudaBuffer::fill(
    const void* element_ptr,
    size_t element_nbytes,
    size_t offset,
    size_t nbytes) {
    COMPAS_ASSERT(element_nbytes > 0 && nbytes % element_nbytes == 0);

    bool all_equal = true;
    for (size_t i = 1; i < element_nbytes; i++) {
        if (static_cast<const char*>(element_ptr)[i] != static_cast<const char*>(element_ptr)[0]) {
            all_equal = false;
        }
    }

    CudaContextGuard guard {context_};
    hipDeviceptr_t ptr = device_ptr_ + offset;

    if (all_equal || element_nbytes == 1) {
        char value = static_cast<const char*>(element_ptr)[0];
        COMPAS_CUDA_CHECK(hipMemsetD8(ptr, value, nbytes));
    } else if (element_nbytes == 2) {
        uint16_t value = static_cast<const uint16_t*>(element_ptr)[0];
        COMPAS_CUDA_CHECK(hipMemsetD16(ptr, value, nbytes));
    } else if (element_nbytes == 4) {
        uint32_t value = static_cast<const uint32_t*>(element_ptr)[0];
        COMPAS_CUDA_CHECK(hipMemsetD32(ptr, value, nbytes));
    } else {
        COMPAS_PANIC("fill can only be performed using 8, 16, or 32 bit values");
    }
}

}  // namespace compas