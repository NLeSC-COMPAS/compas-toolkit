#include "compas/core/assertion.h"
#include "compas/utils/gemm.h"

namespace compas {

void compute_gemm(
    const kmm::DeviceResource& context,
    GPUSubviewMut<cfloat, 2> result,
    GPUSubview<cfloat, 2> lhs,
    GPUSubview<cfloat, 2> rhs,
    cfloat beta,
    GemmComputeMethod kind) {
    hipComplex alpha = {1, 0};

    int64_t m = result.size(0);
    int64_t n = result.size(1);
    int64_t k = lhs.size(1);

    COMPAS_CHECK(result.size(0) == m);
    COMPAS_CHECK(result.size(1) == n);
    COMPAS_CHECK(lhs.size(0) == m);
    COMPAS_CHECK(lhs.size(1) == k);
    COMPAS_CHECK(rhs.size(0) == n);
    COMPAS_CHECK(rhs.size(1) == k);

    hipblasGemmAlgo_t compute_algo = HIPBLAS_GEMM_DEFAULT;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

    switch (kind) {
        case GemmComputeMethod::Pedantic:
            compute_type = HIPBLAS_COMPUTE_32F_PEDANTIC;
            break;
        case GemmComputeMethod::Fast:
            compute_type = HIPBLAS_COMPUTE_32F_PEDANTIC;
            break;
        case GemmComputeMethod::BF16:
            compute_type = HIPBLAS_COMPUTE_32F_FAST_16BF;
            break;
        case GemmComputeMethod::TF32:
            compute_type = HIPBLAS_COMPUTE_32F_FAST_TF32;
            break;
    }

    COMPAS_GPU_CHECK(hipblasSetStream(context.blas(), context.stream()));
    COMPAS_GPU_CHECK(hipblasGemmEx_64(
        context.blas(),
        HIPBLAS_OP_T,  // transa
        HIPBLAS_OP_N,  // transb
        n,  // m
        m,  // n
        k,  // k
        &alpha,  // alpha
        rhs.data(),  // A
        HIP_C_32F,  // A type
        rhs.stride(),  // lda
        lhs.data(),  // B
        HIP_C_32F,  // B type
        lhs.stride(),  // ldb
        &beta,  //beta
        result.data(),  // C
        HIP_C_32F,  // C type
        result.stride(),  // ldc
        compute_type,
        compute_algo));
}

}  // namespace compas
