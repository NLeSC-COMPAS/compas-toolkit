#include <iostream>

#include "core/context.h"
#include "core/utils.h"
#include "parameters/tissue.h"
#include "simulate/signal.h"
#include "simulate/signal_kernels.cuh"
#include "trajectories/spiral.h"

namespace compas {

static void simulate_signal_cartesian(
    const CudaContext& context,
    CudaArray<cfloat, 3> signal,
    CudaArray<cfloat, 2> echos,
    TissueParameters parameters,
    CartesianTrajectory trajectory,
    CudaArray<float, 2> coil_sensitivities) {
    CudaContextGuard guard {context};

    int ncoils = coil_sensitivities.size(0);
    int nvoxels = parameters.nvoxels;
    int nreadouts = trajectory.nreadouts;
    int samples_per_readout = trajectory.samples_per_readout;

    COMPAS_ASSERT(coil_sensitivities.size(1) == nvoxels);

    COMPAS_ASSERT(signal.size(0) == ncoils);
    COMPAS_ASSERT(signal.size(1) == nreadouts);
    COMPAS_ASSERT(signal.size(2) == samples_per_readout);

    COMPAS_ASSERT(echos.size(0) == nreadouts);
    COMPAS_ASSERT(echos.size(1) == nvoxels);

    auto exponents = context.allocate<cfloat, 2>({samples_per_readout, nvoxels});
    auto factors = context.allocate<cfloat>(echos.shape());

    dim3 block_dim = {32, 4};
    dim3 grid_dim = {div_ceil(uint(nvoxels), block_dim.x), div_ceil(uint(nreadouts), block_dim.y)};

    kernels::prepare_signal_factors<<<grid_dim, block_dim>>>(
        factors.view_mut(),
        echos.view(),
        parameters.view(),
        trajectory.view());
    COMPAS_CUDA_CHECK(hipGetLastError());

    block_dim = {256};
    grid_dim = {div_ceil(uint(nvoxels), block_dim.x)};

    kernels::prepare_signal_cartesian<<<grid_dim, block_dim>>>(
        exponents.view_mut(),
        parameters.view(),
        trajectory.view());
    COMPAS_CUDA_CHECK(hipGetLastError());

    const uint threads_per_block = 64;
    const uint threads_cooperative = 32;
    const uint samples_per_thread = 8;
    const uint coils_per_thread = 4;

    block_dim = {threads_per_block};
    grid_dim = {
        div_ceil(
            div_ceil(uint(samples_per_readout), samples_per_thread) * threads_cooperative,
            threads_per_block),
        uint(nreadouts),
        div_ceil(uint(ncoils), uint(coils_per_thread)),
    };

    kernels::sum_signal_cartesian<
        threads_per_block,
        threads_cooperative,
        samples_per_thread,
        coils_per_thread><<<grid_dim, block_dim>>>(
        signal.view_mut(),
        exponents.view(),
        factors.view(),
        coil_sensitivities.view());

    COMPAS_CUDA_CHECK(hipGetLastError());
}

static void simulate_signal_spiral(
    const CudaContext& context,
    CudaArray<cfloat, 3> signal,
    CudaArray<cfloat, 2> echos,
    TissueParameters parameters,
    SpiralTrajectory trajectory,
    CudaArray<float, 2> coil_sensitivities) {
    CudaContextGuard guard {context};

    int ncoils = coil_sensitivities.size(0);
    int nvoxels = parameters.nvoxels;
    int nreadouts = trajectory.nreadouts;
    int samples_per_readout = trajectory.samples_per_readout;

    COMPAS_ASSERT(coil_sensitivities.size(1) == nvoxels);

    COMPAS_ASSERT(signal.size(0) == ncoils);
    COMPAS_ASSERT(signal.size(1) == nreadouts);
    COMPAS_ASSERT(signal.size(2) == samples_per_readout);

    COMPAS_ASSERT(echos.size(0) == nreadouts);
    COMPAS_ASSERT(echos.size(1) == nvoxels);

    auto factors = context.allocate<cfloat>(echos.shape());
    dim3 block_dim = {32, 4};
    dim3 grid_dim = {div_ceil(uint(nvoxels), block_dim.x), div_ceil(uint(nreadouts), block_dim.y)};

    kernels::prepare_signal_factors<<<grid_dim, block_dim>>>(
        factors.view_mut(),
        echos.view(),
        parameters.view(),
        trajectory.view());
    COMPAS_CUDA_CHECK(hipGetLastError());

    auto exponents = context.allocate<cfloat>(echos.shape());
    block_dim = {32, 4};
    grid_dim = {div_ceil(uint(nvoxels), block_dim.x), div_ceil(uint(nreadouts), block_dim.y)};

    kernels::prepare_signal_spiral<<<grid_dim, block_dim>>>(
        exponents.view_mut(),
        parameters.view(),
        trajectory.view());
    COMPAS_CUDA_CHECK(hipGetLastError());

    const uint threads_per_block = 64;
    const uint threads_cooperative = 32;
    const uint samples_per_thread = 8;
    const uint coils_per_thread = 1;

    block_dim = {threads_per_block};
    grid_dim = {
        div_ceil(
            div_ceil(uint(samples_per_readout), samples_per_thread) * threads_cooperative,
            threads_per_block),
        uint(nreadouts),
        div_ceil(uint(ncoils), uint(coils_per_thread)),
    };

    kernels::sum_signal_spiral<
        threads_per_block,
        threads_cooperative,
        samples_per_thread,
        coils_per_thread><<<grid_dim, block_dim>>>(
        signal.view_mut(),
        exponents.view(),
        factors.view(),
        coil_sensitivities.view());

    COMPAS_CUDA_CHECK(hipGetLastError());
}

void simulate_signal(
    const CudaContext& context,
    CudaArray<cfloat, 3> signal,
    CudaArray<cfloat, 2> echos,
    TissueParameters parameters,
    Trajectory trajectory,
    CudaArray<float, 2> coil_sensitivities) {
    if (const auto c = trajectory.as_cartesian()) {
        simulate_signal_cartesian(context, signal, echos, parameters, *c, coil_sensitivities);
    } else if (const auto s = trajectory.as_spiral()) {
        simulate_signal_spiral(context, signal, echos, parameters, *s, coil_sensitivities);
    } else {
        COMPAS_PANIC("invalid trajectory type");
    }
}
}  // namespace compas
