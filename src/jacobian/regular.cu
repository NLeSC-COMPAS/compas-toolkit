#include "hip/hip_runtime.h"
#include "core/utils.h"
#include "jacobian/product.h"

namespace compas {

namespace kernels {

struct DeltaMagnetization {
    cfloat m;
    vec2<cfloat> dm;  // In T1 and T2
};

__device__ DeltaMagnetization delta_to_sample_point(
    cfloat m,
    vec2<cfloat> dm,
    CartesianTrajectoryView trajectory,
    int readout_idx,
    int sample_idx,
    TissueVoxel p) {
    // Read in constants
    auto R2 = 1.0f / p.T2;
    auto ns = trajectory.samples_per_readout;
    auto delta_t = trajectory.delta_t;
    auto delta_k0 = trajectory.delta_k;
    auto x = p.x;
    auto y = p.y;

    // There are ns samples per readout, echo time is assumed to occur
    // at index (ns÷2)+1. Now compute sample index relative to the echo time
    float s = float(sample_idx) - 0.5f * float(ns);

    // Apply readout gradient, T₂ decay and B₀ rotation
    auto Theta = delta_k0.re * x + delta_k0.im * y;
    Theta += delta_t * float(2 * M_PI) * p.B0;
    auto E = exp(s * cfloat(-delta_t * R2, Theta));

    auto dE = vec2<cfloat>(0, (s * delta_t) * R2 * R2 * E);

    auto dms = dm * E + m * dE;
    auto ms = E * m;

    return {ms, dms};
}

__global__ void jacobian_product(
    cuda_view_mut<cfloat> Jv,
    cuda_view<cfloat, 2> echos,
    cuda_view<cfloat, 3> delta_echos,
    TissueParametersView parameters,
    CartesianTrajectoryView trajectory,
    cuda_view<float> coil_sensitivities,
    cuda_view<cfloat, 2> v) {
    auto i = index_t(blockIdx.x * blockDim.x + threadIdx.x);

    int ns = trajectory.samples_per_readout;
    int nr = trajectory.nreadouts;
    auto result = cfloat();

    if (i < nr * ns) {
        int r = i / ns;
        int s = i % ns;
        int nvoxels = parameters.nvoxels;

        for (index_t voxel = 0; voxel < nvoxels; voxel++) {
            // load coordinates, parameters, coil sensitivities and proton density for voxel
            auto p = parameters.get(voxel);
            auto rho = p.rho;

            auto C = coil_sensitivities[voxel];

            // load magnetization and partial derivatives at echo time of the r-th readout
            auto me = echos[r][voxel];
            auto dme = vec2<cfloat> {delta_echos[0][r][voxel], delta_echos[1][r][voxel]};

            // compute decay (T₂) and rotation (gradients and B₀) to go to sample point
            auto [m, dm] = delta_to_sample_point(me, dme, trajectory, r, s, p);

            // store magnetization from this voxel, scaled with v (~ proton density) and C in accumulator
            auto dmv = vec4<cfloat>(v[0][voxel], v[1][voxel], v[2][voxel], v[3][voxel])
                * vec4<cfloat>(dm[0], dm[1], m, m * cfloat(0, 1));

            auto lin_scale = vec4<cfloat>(p.T1 * C * rho, p.T2 * C * rho, C, C);

            result += dot(lin_scale, dmv);
        }

        Jv[i] = result;
    }
}
}  // namespace kernels

void compute_jacobian(
    const CudaContext& ctx,
    cuda_view_mut<cfloat, 2> Jv,
    cuda_view<cfloat, 2> echos,
    cuda_view<cfloat, 3> delta_echos,
    TissueParametersView parameters,
    CartesianTrajectoryView trajectory,
    cuda_view<float, 2> coil_sensitivities,
    cuda_view<cfloat, 2> vector) {
    CudaContextGuard guard {ctx};

    int ns = trajectory.samples_per_readout;
    int nreadouts = trajectory.nreadouts;
    int nvoxels = parameters.nvoxels;
    int ncoils = coil_sensitivities.size(0);

    COMPAS_ASSERT(Jv.size(0) == ncoils);
    COMPAS_ASSERT(Jv.size(1) == nreadouts * ns);
    COMPAS_ASSERT(echos.size(0) == nreadouts);
    COMPAS_ASSERT(echos.size(1) == nvoxels);
    COMPAS_ASSERT(delta_echos.size(0) == 2);  // T1 and T2
    COMPAS_ASSERT(delta_echos.size(1) == nreadouts);
    COMPAS_ASSERT(delta_echos.size(2) == nvoxels);
    COMPAS_ASSERT(coil_sensitivities.size(0) == ncoils);
    COMPAS_ASSERT(coil_sensitivities.size(1) == nvoxels);
    COMPAS_ASSERT(vector.size(0) == 4);  // four reconstruction parameters: T1, T2, rho_x, rho_y
    COMPAS_ASSERT(vector.size(1) == nvoxels);

    dim3 block_dim = 256;
    dim3 grid_dim = div_ceil(uint(nreadouts * ns), block_dim.x);

    // Repeat for each coil
    for (int icoil = 0; icoil < ncoils; icoil++) {
        kernels::jacobian_product<<<grid_dim, block_dim>>>(
            Jv.drop_leading_axis(icoil),
            echos,
            delta_echos,
            parameters,
            trajectory,
            coil_sensitivities.drop_leading_axis(icoil),
            vector);
    }
}
}  // namespace compas