#include <iostream>

#include "compas/core/context.h"
#include "compas/core/utils.h"
#include "compas/trajectories/cartesian.h"
#include "compas/trajectories/signal.h"
#include "compas/trajectories/spiral.h"
#include "signal_kernels.cuh"

namespace compas {

void magnetization_to_signal_cartesian_direct(
    const kmm::DeviceResource& context,
    kmm::Range<index_t> voxels,
    GPUViewMut<cfloat, 3> signal,
    GPUSubview<cfloat, 2> echos,
    TissueParametersView parameters,
    CartesianTrajectoryView trajectory,
    GPUSubview<cfloat, 2> coil_sensitivities,
    GPUSubviewMut<cfloat, 2> sample_decay,
    GPUSubviewMut<cfloat, 2> readout_echos) {
    int voxel_begin = voxels.begin;
    int voxel_end = voxels.end;
    int nvoxels = voxels.size();
    int ncoils = kmm::checked_cast<int>(coil_sensitivities.size(0));
    int nreadouts = trajectory.nreadouts;
    int samples_per_readout = trajectory.samples_per_readout;

    COMPAS_ASSERT(coil_sensitivities.begin(1) <= voxel_begin);
    COMPAS_ASSERT(coil_sensitivities.end(1) >= voxel_end);

    COMPAS_ASSERT(signal.size(0) == ncoils);
    COMPAS_ASSERT(signal.size(1) == nreadouts);
    COMPAS_ASSERT(signal.size(2) == samples_per_readout);

    COMPAS_ASSERT(echos.size(0) == nreadouts);
    COMPAS_ASSERT(echos.begin(1) <= voxel_begin);
    COMPAS_ASSERT(echos.end(1) >= voxel_end);

    dim3 block_dim = {32, 4};
    dim3 grid_dim = {div_ceil(uint(nvoxels), block_dim.x), div_ceil(uint(nreadouts), block_dim.y)};

    kernels::prepare_readout_echos<<<grid_dim, block_dim, 0, context>>>(
        voxels,
        nreadouts,
        readout_echos,
        echos,
        parameters,
        trajectory);
    COMPAS_GPU_CHECK(gpuGetLastError());

    block_dim = {256};
    grid_dim = {div_ceil(uint(nvoxels), block_dim.x)};

    kernels::prepare_sample_decay_cartesian<<<grid_dim, block_dim, 0, context>>>(
        voxels,
        samples_per_readout,
        sample_decay,
        parameters,
        trajectory);
    COMPAS_GPU_CHECK(gpuGetLastError());

    // TODO: Are these faster?
    //    const uint threads_cooperative = 32;
    //    const uint samples_per_thread = 8;
    //    const uint readouts_per_thread = 1;
    //    const uint coils_per_thread = 4;

    const uint block_size_x = 32;
    const uint block_size_y = 8;
    const uint threads_cooperative = 32;
    const uint samples_per_thread = 1;
    const uint readouts_per_thread = 1;
    const uint coils_per_thread = 1;

    block_dim = {block_size_x, block_size_y};
    grid_dim = {
        div_ceil(
            div_ceil(uint(samples_per_readout), samples_per_thread) * threads_cooperative,
            block_size_x),
        div_ceil(uint(nreadouts), readouts_per_thread * block_size_y),
        div_ceil(uint(ncoils), uint(coils_per_thread)),
    };

    kernels::sum_signal_cartesian<
        block_size_x * block_size_y,
        threads_cooperative,
        samples_per_thread,
        readouts_per_thread,
        coils_per_thread><<<grid_dim, block_dim, 0, context>>>(
        voxels,
        signal,
        sample_decay,
        readout_echos,
        coil_sensitivities);

    COMPAS_GPU_CHECK(gpuGetLastError());
    context.synchronize();
}

void magnetization_to_signal_cartesian_gemm(
    const kmm::DeviceResource& context,
    kmm::Range<index_t> voxels,
    GPUViewMut<cfloat, 3> signal,
    GPUView<cfloat, 2> echos,
    TissueParametersView parameters,
    CartesianTrajectoryView trajectory,
    GPUView<cfloat, 2> coil_sensitivities,
    GPUViewMut<cfloat, 2> exponents,
    GPUViewMut<cfloat, 2> factors,
    hipblasComputeType_t compute_type) {
    int ncoils = kmm::checked_cast<int>(coil_sensitivities.size(0));
    int nreadouts = trajectory.nreadouts;
    int nvoxels = voxels.size();
    int samples_per_readout = trajectory.samples_per_readout;

    COMPAS_ASSERT(coil_sensitivities.size(1) == voxels.size());

    COMPAS_ASSERT(signal.size(0) == ncoils);
    COMPAS_ASSERT(signal.size(1) == nreadouts);
    COMPAS_ASSERT(signal.size(2) == samples_per_readout);

    COMPAS_ASSERT(echos.size(0) == nreadouts);
    COMPAS_ASSERT(echos.size(1) == nvoxels);

    dim3 block_dim = {32, 4};
    dim3 grid_dim = {div_ceil(uint(nvoxels), block_dim.x), div_ceil(uint(nreadouts), block_dim.y)};

    kernels::prepare_readout_echos<<<grid_dim, block_dim, 0, context.stream()>>>(
        voxels,
        nreadouts,
        factors,
        echos,
        parameters,
        trajectory);
    COMPAS_GPU_CHECK(gpuGetLastError());

    for (index_t icoil = 0; icoil < ncoils; icoil++) {
        block_dim = {256};
        grid_dim = {div_ceil(uint(nvoxels), block_dim.x)};

        kernels::
            prepare_sample_decay_cartesian_with_coil<<<grid_dim, block_dim, 0, context.stream()>>>(
                exponents,
                coil_sensitivities.drop_axis<0>(icoil),
                parameters,
                trajectory);
        COMPAS_GPU_CHECK(gpuGetLastError());

        hipComplex alpha = {1, 0};
        hipComplex beta = {0, 0};

        hipDataType output_type = HIP_C_32F;
        hipDataType input_type = HIP_C_32F;
        hipblasGemmAlgo_t compute_algo = HIPBLAS_GEMM_DEFAULT;

        COMPAS_GPU_CHECK(hipblasSetStream(context.blas(), context.stream()));
        COMPAS_GPU_CHECK(hipblasGemmEx(
            context.blas(),
            HIPBLAS_OP_T,  // transa
            HIPBLAS_OP_N,  // transb
            samples_per_readout,  // m
            nreadouts,  // n
            nvoxels,  // k
            &alpha,  // alpha
            exponents.data(),  // A
            input_type,  // A type
            nvoxels,  // lda
            factors.data(),  // B
            input_type,  // B type
            nvoxels,  // ldb
            &beta,  //beta
            signal.data() + signal.stride(0) * icoil,  // C
            output_type,  // C type
            samples_per_readout,  // ldc
            compute_type,
            compute_algo));
    }

    COMPAS_GPU_CHECK(gpuGetLastError());
}

void magnetization_to_signal_spiral(
    const kmm::DeviceResource& context,
    kmm::Range<index_t> voxels,
    GPUViewMut<cfloat, 3> signal,
    GPUView<cfloat, 2> echos,
    TissueParametersView parameters,
    SpiralTrajectoryView trajectory,
    GPUView<cfloat, 2> coil_sensitivities,
    GPUViewMut<cfloat, 2> sample_decay,
    GPUViewMut<cfloat, 2> readout_echos) {
    int ncoils = kmm::checked_cast<int>(coil_sensitivities.size(0));
    int nreadouts = trajectory.nreadouts;
    int samples_per_readout = trajectory.samples_per_readout;

    COMPAS_ASSERT(coil_sensitivities.size(1) == voxels.size());

    COMPAS_ASSERT(signal.size(0) == ncoils);
    COMPAS_ASSERT(signal.size(1) == nreadouts);
    COMPAS_ASSERT(signal.size(2) == samples_per_readout);

    COMPAS_ASSERT(echos.size(0) == nreadouts);
    COMPAS_ASSERT(echos.size(1) == voxels.size());

    dim3 block_dim = {32, 4};
    dim3 grid_dim = {
        div_ceil(uint(voxels.size()), block_dim.x),
        div_ceil(uint(nreadouts), block_dim.y)};

    kernels::prepare_readout_echos<<<grid_dim, block_dim, 0, context.stream()>>>(
        voxels,
        nreadouts,
        readout_echos,
        echos,
        parameters,
        trajectory);
    COMPAS_GPU_CHECK(gpuGetLastError());

    block_dim = {32, 4};
    grid_dim = {div_ceil(uint(voxels.size()), block_dim.x), div_ceil(uint(nreadouts), block_dim.y)};

    kernels::prepare_sample_decay_spiral<<<grid_dim, block_dim, 0, context.stream()>>>(
        sample_decay,
        parameters,
        trajectory);
    COMPAS_GPU_CHECK(gpuGetLastError());

    const uint threads_per_block = 64;
    const uint threads_cooperative = 32;
    const uint samples_per_thread = 8;
    const uint coils_per_thread = 1;

    block_dim = {threads_per_block};
    grid_dim = {
        div_ceil(
            div_ceil(uint(samples_per_readout), samples_per_thread) * threads_cooperative,
            threads_per_block),
        uint(nreadouts),
        div_ceil(uint(ncoils), uint(coils_per_thread)),
    };

    kernels::sum_signal_spiral<
        threads_per_block,
        threads_cooperative,
        samples_per_thread,
        coils_per_thread><<<grid_dim, block_dim, 0, context.stream()>>>(
        signal,
        sample_decay,
        readout_echos,
        coil_sensitivities);

    COMPAS_GPU_CHECK(gpuGetLastError());
}

hipblasComputeType_t cublas_compute_type_from_simulate_method(SimulateSignalMethod method) {
    switch (method) {
        case SimulateSignalMethod::MatmulPedantic:
            return HIPBLAS_COMPUTE_32F_PEDANTIC;
        case SimulateSignalMethod::Matmul:
            return HIPBLAS_COMPUTE_32F;
        case SimulateSignalMethod::MatmulBF16:
            return HIPBLAS_COMPUTE_32F_FAST_16BF;
        case SimulateSignalMethod::MatmulTF32:
            return HIPBLAS_COMPUTE_32F_FAST_TF32;
        default:
            COMPAS_PANIC("invalid value for `SimulateSignalMethod`");
    }
}

Array<cfloat, 3> magnetization_to_signal(
    const CompasContext& context,
    Array<cfloat, 2> echos,
    TissueParameters parameters,
    const Trajectory& trajectory,
    Array<cfloat, 2> coil_sensitivities,
    SimulateSignalMethod method) {
    using namespace kmm::placeholders;

    int ncoils = kmm::checked_cast<int>(coil_sensitivities.size(0));
    int nvoxels = parameters.nvoxels;
    int chunk_size = parameters.chunk_size;
    int nreadouts = trajectory.nreadouts;
    int samples_per_readout = trajectory.samples_per_readout;

    auto signal = Array<cfloat, 3> {{ncoils, nreadouts, samples_per_readout}};

    if (const auto* cart = dynamic_cast<const CartesianTrajectory*>(&trajectory)) {
        auto temp_exponents = Array<cfloat, 2> {{samples_per_readout, nvoxels}};
        auto temp_factors = Array<cfloat, 2> {{nreadouts, nvoxels}};

        if (method == SimulateSignalMethod::Naive) {
            context.submit_kernel(
                {uint(samples_per_readout), uint(nreadouts), uint(ncoils)},
                256,
                kernels::sum_signal_cartesian_naive,
                nvoxels,
                write(signal),
                echos,
                parameters,
                *cart,
                coil_sensitivities);

        } else if (method == SimulateSignalMethod::Direct) {
            context.parallel_device(
                {nvoxels, nreadouts},
                {chunk_size, nreadouts},
                magnetization_to_signal_cartesian_direct,
                _x,
                reduce(kmm::Reduction::Sum, signal),
                echos[_y][_x],
                parameters.data[_][_x],
                *cart,
                coil_sensitivities[_][_x],
                write(temp_exponents[_][_x]),
                write(temp_factors[_y][_x]));
        } else {
            context.submit_device(
                magnetization_to_signal_cartesian_gemm,
                nvoxels,
                write(signal),
                echos,
                parameters,
                *cart,
                coil_sensitivities,
                write(temp_exponents),
                write(temp_factors),
                cublas_compute_type_from_simulate_method(method));
        }
    } else if (const auto* s = dynamic_cast<const SpiralTrajectory*>(&trajectory)) {
        auto temp_exponents = Array<cfloat, 2>(echos.shape());
        auto temp_factors = Array<cfloat, 2>(echos.shape());

        context.submit_device(
            magnetization_to_signal_spiral,
            _x,
            write(signal),
            echos,
            parameters,
            *s,
            coil_sensitivities,
            write(temp_exponents),
            write(temp_factors));
    } else {
        COMPAS_PANIC("invalid trajectory type");
    }

    return signal;
}
}  // namespace compas
